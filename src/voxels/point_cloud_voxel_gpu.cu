#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/driver_types.h>
#include <chrono>


#include "voxels/point_cloud/point_cloud_voxel_gpu.h"
// Linker issue caused here
// #include "spdlog/spdlog.h"

#define MIN(a, b) (a < b ? a : b)
#define MAX(a, b) (a < b ? b : a)
#define CEIL(x, y) ((x + y - 1) / y)

#define MAX_THREADS_PER_BLOCK 1024
#define THREADS_PER_WARP 32
#define WARPS_PER_BLOCK 32
#define WARP_SIZE 32
#define FLOATS_PER_POINT 3
#define NUM_PCV1_POINTS_PER_THREAD 1
#define NUM_PCV2_POINTS_PER_THREAD 512


using namespace grrt;

// TODO: siddharth: add cuda kernels

// each thread works on 3 floats from pcl_voxel_1 and 3 floats from pcl_voxel_2
__global__ void saxby_shuffle_single(float* pcl_voxel_1_pnts,  float* pcl_voxel_2_pnts, int pcl_voxel_1_count, int pcl_voxel_2_count, float* bool_sum, int bool_sum_size) {

    int warp_id = threadIdx.x / THREADS_PER_WARP;
    int lane_id = threadIdx.x % THREADS_PER_WARP;

    // A thread acts on three floats
    int pcl_v1_start_i = (blockIdx.x * MAX_THREADS_PER_BLOCK * NUM_PCV1_POINTS_PER_THREAD + lane_id * WARP_SIZE + warp_id) * FLOATS_PER_POINT;
    int pcl_v2_start_i = blockIdx.y * FLOATS_PER_POINT * NUM_PCV2_POINTS_PER_THREAD;

    // if (pcl_v1_start_i >= pcl_voxel_1_count || pcl_v2_start_i >= pcl_voxel_2_count) {
    //     return;
    // }

    if (pcl_v1_start_i + (NUM_PCV1_POINTS_PER_THREAD * FLOATS_PER_POINT) - 1 >= pcl_voxel_1_count 
    || pcl_v2_start_i + (NUM_PCV2_POINTS_PER_THREAD * FLOATS_PER_POINT) - 1 >=  pcl_voxel_2_count) {
        return;
    }

    // printf("pcl_v1 point: %d\n", pcl_v1_start_i);
    // printf("pcl_v2 point: %d\n", pcl_v2_i);

    int sum = 0;
    
    for (int pcl_v1_i = pcl_v1_start_i; pcl_v1_i < pcl_v1_start_i + NUM_PCV1_POINTS_PER_THREAD * FLOATS_PER_POINT; pcl_v1_i += 3) {
        // printf("Point: (%f, %f, %f) at %d and Point: (%f, %f, %f) at %d\n", pcl_voxel_1_pnts[pcl_v1_i], pcl_voxel_1_pnts[pcl_v1_i + 1], pcl_voxel_1_pnts[pcl_v1_i + 2], pcl_v1_i, pcl_voxel_2_pnts[pcl_v2_i], pcl_voxel_2_pnts[pcl_v2_i + 1], pcl_voxel_2_pnts[pcl_v2_i + 2], pcl_v2_i);

        for (int pcl_v2_i = pcl_v2_start_i; pcl_v2_i < pcl_v2_start_i + NUM_PCV2_POINTS_PER_THREAD * FLOATS_PER_POINT; pcl_v2_i += 3) {

        float dist = std::sqrt(
        (pcl_voxel_1_pnts[pcl_v1_i] - pcl_voxel_2_pnts[pcl_v2_i]) * (pcl_voxel_1_pnts[pcl_v1_i] - pcl_voxel_2_pnts[pcl_v2_i]) 
        + (pcl_voxel_1_pnts[pcl_v1_i + 1] - pcl_voxel_2_pnts[pcl_v2_i + 1]) * (pcl_voxel_1_pnts[pcl_v1_i + 1] - pcl_voxel_2_pnts[pcl_v2_i + 1]) 
        + (pcl_voxel_1_pnts[pcl_v1_i + 2] - pcl_voxel_2_pnts[pcl_v2_i + 2]) * (pcl_voxel_1_pnts[pcl_v1_i + 2] - pcl_voxel_2_pnts[pcl_v2_i + 2]));

        // printf("dist: %f\n", dist);

        // if (dist < 2) {
        //     printf("Point: (%f, %f, %f) and Point: (%f, %f, %f) are within 0.15\n", pcl_voxel_1_pnts[pcl_v1_i], pcl_voxel_1_pnts[pcl_v1_i + 1], pcl_voxel_1_pnts[pcl_v1_i + 2], pcl_voxel_2_pnts[pcl_v2_i], pcl_voxel_2_pnts[pcl_v2_i + 1], pcl_voxel_2_pnts[pcl_v2_i + 2]);
        // }

        if (dist < PCL_VOXEL_RADIUS) {
            sum += 1;
        }
        }
    }

    __syncthreads();

    for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1)
        sum += __shfl_down_sync(0xffffff, sum, offset);

    if (lane_id == 0) {
        int bool_i = blockIdx.x * WARPS_PER_BLOCK + warp_id;

        // printf("bool_i: %d\n", bool_i);

        if (bool_i >= bool_sum_size) {
            printf("invalid acccess int bool_sum_size\n");
        }
        bool_sum[bool_i] += sum;
    }
}


PointCloudVoxelGPU::PointCloudVoxelGPU(const size_t num_points) : num_points(num_points) {
    hipError_t err = hipHostAlloc(&points, num_points * sizeof(Point), hipHostMallocDefault);
    if (err != hipSuccess) {
        printf("Failed to allocate memory for point cloud voxel: %s", hipGetErrorString(err));
    }
}

PointCloudVoxelGPU::~PointCloudVoxelGPU(){
    hipError_t err = hipHostFree(this->points);
    if (err != hipSuccess) {
        printf("Failed to free points memory for point cloud voxel gpu: %s", hipGetErrorString(err));
    }
}


void PointCloudVoxelGPU::addPoint(const Point& point) {
    if (this->current_num_points >= this->num_points) {
        printf("Point cloud voxel GPU is full\n");
        return;
    }
    this->points[this->current_num_points] = point.x;
    this->points[this->current_num_points + 1] = point.y;
    this->points[this->current_num_points + 2] = point.z;
    this->current_num_points += 3;
}

bool PointCloudVoxelGPU::contains(const Point& point) const {
    // TODO
    return false;
}

bool PointCloudVoxelGPUManager::intersect(const Voxel::SharedPtr& voxel_1, const Voxel::SharedPtr& voxel_2)  {
    // will have to cast the vortex to a PointCloudVoxelGPU.
    PointCloudVoxelGPU::SharedPtr pcl_voxel_1 = std::dynamic_pointer_cast<PointCloudVoxelGPU>(voxel_1);
    PointCloudVoxelGPU::SharedPtr pcl_voxel_2 = std::dynamic_pointer_cast<PointCloudVoxelGPU>(voxel_2);

    float *bool_sum;

    // Each MAX_THREADS_PER_BLOCK * NUM_PCV1_POINTS_PER_THREAD Points in pcl_voxel_1 will be worked on by a block, but the shuffle operation only occurs within a warp which is 32 threads aka 32 points (through 32 * 3 floats) in pcv1.
    printf("blah: %lu\n", pcl_voxel_1->num_points);
    int num_blocks_pcv1 = CEIL(pcl_voxel_1->num_points, (3 * MAX_THREADS_PER_BLOCK * NUM_PCV1_POINTS_PER_THREAD));

    // int bool_sum_size = CEIL(pcl_voxel_1->num_points, (3 * MAX_THREADS_PER_BLOCK * NUM_PCV1_POINTS_PER_THREAD / WARP_SIZE));
    int bool_sum_size = num_blocks_pcv1 * WARP_SIZE;

    hipError_t err = hipHostAlloc(&bool_sum, bool_sum_size * sizeof(float), hipHostMallocDefault);
    if (err != hipSuccess) {
        printf("Failed to allocate memory for bool_sum");
    }

    printf("bool_sum_size: %d\n", bool_sum_size);

    int num_blocks_pcv2 = CEIL(pcl_voxel_2->num_points, (3 * NUM_PCV2_POINTS_PER_THREAD));


    auto start = std::chrono::high_resolution_clock::now();

    // 30 elements from voxel_1 and voxel_2 will be assigned to a block (we won't use the full 32 since we need to operate over 3 floats at a time)


    // One block runs over MAX_THREADS_PER_BLOCK * NUM_PCV1_POINTS_PER_THREAD Points in pcl_voxel_1 and one Point in pcl_voxel_2
    dim3 num_blocks(num_blocks_pcv1, num_blocks_pcv2, 1);

    // printf("pcl_voxel_2 points: %f\n", pcl_voxel_2->points[0]);
    // printf("pcl_voxel_2 points: %f\n", pcl_voxel_2->points[1]);
    // printf("pcl_voxel_2 points: %f\n", pcl_voxel_2->points[2]);

    printf("a: %d\n", num_blocks_pcv1);
    printf("b: %d\n", num_blocks_pcv2);

    dim3 max_threads_per_block(MAX_THREADS_PER_BLOCK, 1, 1);
    saxby_shuffle_single<<<num_blocks, max_threads_per_block>>>(pcl_voxel_1->points, pcl_voxel_2->points, pcl_voxel_1->num_points, pcl_voxel_2->num_points, bool_sum, bool_sum_size);
    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();

    err = hipHostFree(bool_sum);
    if (err != hipSuccess) {
        printf("Failed to free bool_sum for point cloud voxel gpu\n");
    }

    auto diff = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

    for (int i = 0; i < bool_sum_size; i++) {
        if (bool_sum[i] > 0) {
            printf("collision detected\n");
            return true;
        }
    }

     return false;
}