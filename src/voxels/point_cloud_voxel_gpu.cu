#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/driver_types.h>

#include "voxels/point_cloud/point_cloud_voxel_gpu.h"

using namespace grrt;

// TODO: siddharth: add cuda kernels

PointCloudVoxelGPU::PointCloudVoxelGPU(const size_t num_points) : num_points(num_points) {
    hipError_t err = hipHostAlloc(&points, num_points * sizeof(Point), hipHostMallocDefault);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to allocate memory for point cloud voxel");
    }
}

void PointCloudVoxelGPU::addPoint(const Point& point) {}

bool PointCloudVoxelGPU::contains(const Point& point) const {}
