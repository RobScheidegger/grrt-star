#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/driver_types.h>
#include <chrono>

#include "constants.h"
#include "voxels/point_cloud/point_cloud_voxel_gpu.h"

#define MIN(a, b) (a < b ? a : b)
#define MAX(a, b) (a < b ? b : a)
#define CEIL(x, y) ((x + y - 1) / y)

#define MAX_THREADS_PER_BLOCK 1024
#define THREADS_PER_WARP 32
#define WARPS_PER_BLOCK 32
#define WARP_SIZE 32
#define FLOATS_PER_POINT 3
#define NUM_PCV1_POINTS_PER_THREAD 1
#define NUM_PCV2_POINTS_PER_THREAD 512

using namespace grrt;

#define checkCudaErr(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

// each thread works on 3 floats from pcl_voxel_1 and 3 floats from pcl_voxel_2
__global__ void intersect_shuffle(float* pcl_voxel_1_pnts, float* pcl_voxel_2_pnts, int pcl_voxel_1_count,
                                  int pcl_voxel_2_count, int* bool_sum, int bool_sum_size) {

    const int warp_id = threadIdx.x / THREADS_PER_WARP;
    const int lane_id = threadIdx.x % THREADS_PER_WARP;

    // A thread acts on three floats
    const int pcl_v1_start_i = (blockIdx.x * MAX_THREADS_PER_BLOCK + lane_id * WARP_SIZE + warp_id) * FLOATS_PER_POINT *
                               NUM_PCV1_POINTS_PER_THREAD;
    const int pcl_v2_start_i = blockIdx.y * FLOATS_PER_POINT * NUM_PCV2_POINTS_PER_THREAD;

    if (pcl_v1_start_i >= pcl_voxel_1_count || pcl_v2_start_i >= pcl_voxel_2_count) {
        return;
    }

    int sum = 0;

    for (int pcl_v1_i = pcl_v1_start_i; pcl_v1_i < pcl_v1_start_i + NUM_PCV1_POINTS_PER_THREAD * FLOATS_PER_POINT;
         pcl_v1_i += 3) {

        if (sum)
            break;

        if (pcl_v1_i >= pcl_voxel_1_count) {
            return;
        }

        for (int pcl_v2_i = pcl_v2_start_i; pcl_v2_i < pcl_v2_start_i + NUM_PCV2_POINTS_PER_THREAD * FLOATS_PER_POINT;
             pcl_v2_i += 3) {

            if (pcl_v2_i >= pcl_voxel_2_count) {
                break;
            }
            const float dist = (pcl_voxel_1_pnts[pcl_v1_i] - pcl_voxel_2_pnts[pcl_v2_i]) *
                                   (pcl_voxel_1_pnts[pcl_v1_i] - pcl_voxel_2_pnts[pcl_v2_i]) +
                               (pcl_voxel_1_pnts[pcl_v1_i + 1] - pcl_voxel_2_pnts[pcl_v2_i + 1]) *
                                   (pcl_voxel_1_pnts[pcl_v1_i + 1] - pcl_voxel_2_pnts[pcl_v2_i + 1]) +
                               (pcl_voxel_1_pnts[pcl_v1_i + 2] - pcl_voxel_2_pnts[pcl_v2_i + 2]) *
                                   (pcl_voxel_1_pnts[pcl_v1_i + 2] - pcl_voxel_2_pnts[pcl_v2_i + 2]);

            if (dist < PCL_VOXEL_RADIUS * PCL_VOXEL_RADIUS) {
                sum += 1;
                break;
            }
        }
    }

    __syncthreads();

    for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1)
        sum += __shfl_down_sync(0xffffff, sum, offset);

    if (lane_id == 0) {
        int bool_i = blockIdx.x * WARPS_PER_BLOCK + warp_id;

        if (bool_i >= bool_sum_size) {
            printf("invalid acccess int bool_sum_size\n");
        }
        bool_sum[bool_i] += sum;
    }
}

PointCloudVoxelGPU::PointCloudVoxelGPU(const size_t num_points) : num_points(num_points) {
    hipError_t err = hipHostAlloc(&points, num_points * sizeof(Point), hipHostMallocDefault);
    if (err != hipSuccess) {
        printf("Failed to allocate memory for point cloud voxel: %s", hipGetErrorString(err));
    }
}

PointCloudVoxelGPU::~PointCloudVoxelGPU() {
    hipError_t err = hipHostFree(this->points);
    if (err != hipSuccess) {
        printf("Failed to free points memory for point cloud voxel gpu: %s", hipGetErrorString(err));
    }
}

void PointCloudVoxelGPU::addPoint(const Point& point) {
    if (this->current_num_points >= this->num_points) {
        printf("Point cloud voxel GPU is full: %lu\n", num_points);
        return;
    }
    this->points[this->current_num_points] = point.x;
    this->points[this->current_num_points + 1] = point.y;
    this->points[this->current_num_points + 2] = point.z;
    this->current_num_points += 3;
}

bool PointCloudVoxelGPU::contains(const Point& point) const {
    // TODO
    return false;
}

bool PointCloudVoxelGPUManager::intersect(const Voxel::SharedPtr& voxel_1, const Voxel::SharedPtr& voxel_2) {
    // will have to cast the vortex to a PointCloudVoxelGPU.
    PointCloudVoxelGPU::SharedPtr pcl_voxel_1 = std::dynamic_pointer_cast<PointCloudVoxelGPU>(voxel_1);
    PointCloudVoxelGPU::SharedPtr pcl_voxel_2 = std::dynamic_pointer_cast<PointCloudVoxelGPU>(voxel_2);

    int num_blocks_pcv1 = CEIL(pcl_voxel_1->num_points, (3 * MAX_THREADS_PER_BLOCK * NUM_PCV1_POINTS_PER_THREAD));

    int bool_sum_size = num_blocks_pcv1 * WARP_SIZE;

    if (bool_sum_size > BOOL_SUM_MAX_SIZE) {
        printf("bool_sum doesn't have any space for this intersection\n");
    }

    memset(bool_sum, 0, bool_sum_size * sizeof(float));

    int num_blocks_pcv2 = CEIL(pcl_voxel_2->num_points, (3 * NUM_PCV2_POINTS_PER_THREAD));
    dim3 num_blocks(num_blocks_pcv1, num_blocks_pcv2, 1);

    dim3 max_threads_per_block(MAX_THREADS_PER_BLOCK, 1, 1);
    intersect_shuffle<<<num_blocks, max_threads_per_block>>>(pcl_voxel_1->points, pcl_voxel_2->points,
                                                             pcl_voxel_1->num_points, pcl_voxel_2->num_points, bool_sum,
                                                             bool_sum_size);

    checkCudaErr(hipPeekAtLastError());

    checkCudaErr(hipDeviceSynchronize());

    bool res = false;

    for (int i = 0; i < bool_sum_size; i++) {
        if (bool_sum[i] > 0) {
            res = true;
            break;
        }
    }

    return res;
}

PointCloudVoxelGPUManager::PointCloudVoxelGPUManager() {
    hipError_t err = hipHostAlloc(&bool_sum, BOOL_SUM_MAX_SIZE * sizeof(int), hipHostMallocDefault);
    if (err != hipSuccess) {
        printf("Failed to allocate memory for bool_sum: %s", hipGetErrorString(err));
    }
}

PointCloudVoxelGPUManager::~PointCloudVoxelGPUManager() {
    if (bool_sum == nullptr) {
        return;
    }
    hipError_t err = hipHostFree(bool_sum);
    if (err != hipSuccess) {
        printf("Failed to free bool_sum for point cloud voxel gpu\n");
    }
}