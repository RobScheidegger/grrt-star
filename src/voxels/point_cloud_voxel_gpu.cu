#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/driver_types.h>


#include "voxels/point_cloud/point_cloud_voxel_gpu.h"
// Linker issue caused here
// #include "spdlog/spdlog.h"


using namespace grrt;

// TODO: siddharth: add cuda kernels

PointCloudVoxelGPU::PointCloudVoxelGPU(const size_t num_points) : num_points(num_points) {
    // TODO: add a destructor
    hipError_t err = hipHostAlloc(&points, num_points * sizeof(Point), hipHostMallocDefault);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to allocate memory for point cloud voxel");
    }
}

// TODO: create PointCloudVoxelManager but for gpu with the intersect function

void PointCloudVoxelGPU::addPoint(const Point& point) {
    if (this->current_num_points >= this->num_points) {
        // spdlog::error("Point cloud voxel GPU is full");
    }
    this->points[this->current_num_points] = point.x;
    this->points[this->current_num_points + 1] = point.y;
    this->points[this->current_num_points + 2] = point.z;
    this->current_num_points += 3;
}

bool PointCloudVoxelGPU::contains(const Point& point) const {
    // TODO
    return false;
}
