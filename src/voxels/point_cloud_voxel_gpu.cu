#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/driver_types.h>
#include <chrono>


#include "voxels/point_cloud/point_cloud_voxel_gpu.h"
// Linker issue caused here
// #include "spdlog/spdlog.h"

#define MIN(a, b) (a < b ? a : b)
#define MAX(a, b) (a < b ? b : a)
#define CEIL(x, y) ((x + y - 1) / y)

#define MAX_THREADS_PER_BLOCK 1024
#define THREADS_PER_WARP 32
#define WARPS_PER_BLOCK 32
#define WARP_SIZE 32
#define FLOATS_PER_POINT 3


using namespace grrt;

// TODO: siddharth: add cuda kernels

// each thread works on 3 floats from pcl_voxel_1 and 3 floats from pcl_voxel_2
__global__ void saxby_shuffle_single(float* pcl_voxel_1_pnts,  float* pcl_voxel_2_pnts, int pcl_voxel_1_count, int pcl_voxel_2_count, float* bool_sum) {

    int warp_id = threadIdx.x / THREADS_PER_WARP;
    int lane_id = threadIdx.x % THREADS_PER_WARP;

    // A thread acts on three floats
    int pcl_v1_i = blockIdx.x * THREADS_PER_WARP * FLOATS_PER_POINT + warp_id;
    int pcl_v2_i = blockIdx.y * FLOATS_PER_POINT;

    if (pcl_v1_i >= pcl_voxel_1_count || pcl_v2_i >= pcl_voxel_2_count) {
        return;
    }
    
    int sum = 0;

    float dist = std::sqrt(
        (pcl_voxel_1_pnts[pcl_v1_i] - pcl_voxel_2_pnts[pcl_v2_i]) * (pcl_voxel_1_pnts[pcl_v1_i] - pcl_voxel_2_pnts[pcl_v2_i]) 
        + (pcl_voxel_1_pnts[pcl_v1_i + 1] - pcl_voxel_2_pnts[pcl_v2_i + 1]) * (pcl_voxel_1_pnts[pcl_v1_i + 1] - pcl_voxel_2_pnts[pcl_v2_i + 1]) 
        + (pcl_voxel_1_pnts[pcl_v1_i + 2] - pcl_voxel_2_pnts[pcl_v2_i + 2]) * (pcl_voxel_1_pnts[pcl_v1_i + 2] - pcl_voxel_2_pnts[pcl_v2_i + 2]));

    if (dist < PCL_VOXEL_RADIUS) {
        sum = 1;
    }

    __syncthreads();

    for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1)
        sum += __shfl_down_sync(0xffffff, sum, offset);

    if (lane_id == 0) {
        bool_sum[CEIL(pcl_v1_i, FLOATS_PER_POINT)] += sum;
    }
}


PointCloudVoxelGPU::PointCloudVoxelGPU(const size_t num_points) : num_points(num_points) {
    hipError_t err = hipHostAlloc(&points, num_points * sizeof(Point), hipHostMallocDefault);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to allocate memory for point cloud voxel");
    }
}

PointCloudVoxelGPU::~PointCloudVoxelGPU(){
    hipError_t err = hipHostFree(points);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to free points memory for point cloud voxel gpu");
    }
}


void PointCloudVoxelGPU::addPoint(const Point& point) {
    if (this->current_num_points >= this->num_points) {
        throw std::runtime_error("Point cloud voxel GPU is full");
    }
    this->points[this->current_num_points] = point.x;
    this->points[this->current_num_points + 1] = point.y;
    this->points[this->current_num_points + 2] = point.z;
    this->current_num_points += 3;
}

bool PointCloudVoxelGPU::contains(const Point& point) const {
    // TODO
    return false;
}

bool intersect(const Voxel::SharedPtr& voxel_1, const Voxel::SharedPtr& voxel_2)  {
    // will have to cast the vortex to a PointCloudVoxelGPU.
    PointCloudVoxelGPU::SharedPtr pcl_voxel_1 = std::dynamic_pointer_cast<PointCloudVoxelGPU>(voxel_1);
    PointCloudVoxelGPU::SharedPtr pcl_voxel_2 = std::dynamic_pointer_cast<PointCloudVoxelGPU>(voxel_2);

    float *bool_sum;

    // Each MAX_THREADS_PER_BLOCK Points in pcl_voxel_1 will be worked on by a block, but the shuffle operation only occurs within a warp which is 32 threads aka 32 points (through 32 * 3 floats) in pcv1.
    int num_blocks_pcv1 = CEIL(pcl_voxel_1->num_points, 3 * MAX_THREADS_PER_BLOCK);

    hipError_t err = hipHostAlloc(&bool_sum, num_blocks_pcv1 * WARP_SIZE * sizeof(float), hipHostMallocDefault);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to allocate memory for bool_sum");
    }

    auto start = std::chrono::high_resolution_clock::now();

    // 30 elements from voxel_1 and voxel_2 will be assigned to a block (we won't use the full 32 since we need to operate over 3 floats at a time)


    // One block runs over MAX_THREADS_PER_BLOCK Points in pcl_voxel_1 and one Point in pcl_voxel_2
    dim3 num_blocks(num_blocks_pcv1, CEIL(pcl_voxel_2->num_points, 3), 1);
    dim3 max_threads_per_block(MAX_THREADS_PER_BLOCK, 1, 1);
    saxby_shuffle_single<<<num_blocks, max_threads_per_block>>>(pcl_voxel_1->points, pcl_voxel_2->points, pcl_voxel_1->num_points, pcl_voxel_2->num_points, bool_sum);
    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();

    err = hipHostFree(bool_sum);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to free bool_sum for point cloud voxel gpu");
    }

    return std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
}